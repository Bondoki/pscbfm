#include "hip/hip_runtime.h"
/*
 * UpdaterGPUScBFM_AB_Type.cpp
 *
 *  Created on: 27.07.2017
 *      Author: Ron Dockhorn
 */


#include <cstdio>                           // printf
#include <cstdlib>                          // exit
#include <cstring>                          // memset
#include <ctime>
#include <iostream>
#include <stdexcept>
#include <stdint.h>
#include <sstream>

#include "UpdaterGPUScBFM_AB_Type.h"

#define DEBUG_UPDATERGPUSCBFM_AB_TYPE 100


/* why 512??? Because 512==8^3 ??? but that would mean 8 possible values instead of
 * -4 to +4 which I saw being used ... */
__device__ __constant__ bool dpForbiddenBonds[512]; //false-allowed; true-forbidden

/**
 * These will be initialized to:
 *   DXTable_d = { -1,1,0,0,0,0 }
 *   DYTable_d = { 0,0,-1,1,0,0 }
 *   DZTable_d = { 0,0,0,0,-1,1 }
 * I.e. a table of three random directional 3D vectors \vec{dr} = (dx,dy,dz)
 */
__device__ __constant__ intCUDA DXTable_d[6]; //0:-x; 1:+x; 2:-y; 3:+y; 4:-z; 5+z
__device__ __constant__ intCUDA DYTable_d[6]; //0:-x; 1:+x; 2:-y; 3:+y; 4:-z; 5+z
__device__ __constant__ intCUDA DZTable_d[6]; //0:-x; 1:+x; 2:-y; 3:+y; 4:-z; 5+z

/* will this really bring performance improvement? At least constant cache
 * might be as fast as register access when all threads in a warp access the
 * the same constant */
__device__ __constant__ uint32_t dcBoxXM1  ;  // mLattice size in X-1
__device__ __constant__ uint32_t dcBoxYM1  ;  // mLattice size in Y-1
__device__ __constant__ uint32_t dcBoxZM1  ;  // mLattice size in Z-1

__device__ __constant__ uint32_t dcBoxXLog2 ;  // mLattice shift in X
__device__ __constant__ uint32_t dcBoxXYLog2;  // mLattice shift in X*Y

/* Since CUDA 5.5 (~2014) there do exist texture objects which are much
 * easier and can actually be used as kernel arguments!
 * @see https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/
 * "What is not commonly known is that each outstanding texture reference that
 *  is bound when a kernel is launched incurs added launch latency—up to 0.5 μs
 *  per texture reference. This launch overhead persists even if the outstanding
 *  bound textures are not even referenced by the kernel. Again, using texture
 *  objects instead of texture references completely removes this overhead."
 * -> wow !!!
 */
/**
 * Contains the particles as well as a property tag for each:
 *   [ x0, y0, z0, p0, x1, y1, z1, p1, ... ]
 * The propertie tags p are bit packed:
 *                        8  7  6  5  4  3  2  1  0
 * +--------+--+--+--+--+--+--+--+--+--+--+--+--+--+
 * | unused |  |  |  |  |c |   nnr  |  dir   |move |
 * +--------+--+--+--+--+--+--+--+--+--+--+--+--+--+
 *  c   ... charged: 0 no, 1: yes
 *  nnr ... number of neighbors, this will get populated from LeMonADE's
 *          get get
 */
texture< intCUDA, hipTextureType1D, hipReadModeElementType > texPolymerAndMonomerIsEvenAndOnXRef;

hipTextureObject_t texLatticeRefOut = 0;
hipTextureObject_t texLatticeTmpRef = 0;

/**
 * These are arrays containing the monomer indices for the respective
 * species (sorted ascending). E.g. for AABABBA this would be:
 * texSpeciesIndicesA = { 0,1,3,6 }
 * texSpeciesIndicesB = { 1,4,5 }
 */
hipTextureObject_t texSpeciesIndicesA = 0;
hipTextureObject_t texSpeciesIndicesB = 0;



__device__ uint32_t hash( uint32_t a )
{
    /* https://web.archive.org/web/20120626084524/http://www.concentric.net:80/~ttwang/tech/inthash.htm
     * Note that before this 2007-03 version there were no magic numbers.
     * This hash function doesn't seem to be published.
     * He writes himself that this shouldn't really be used for PRNGs ???
     * @todo E.g. check random distribution of randomly drawn directions are
     *       they rouhgly even?
     * The 'hash' or at least an older version of it can even be inverted !!!
     * http://c42f.github.io/2015/09/21/inverting-32-bit-wang-hash.html
     * Somehow this also gets attibuted to Robert Jenkins?
     * https://gist.github.com/badboy/6267743
     * -> http://www.burtleburtle.net/bob/hash/doobs.html
     *    http://burtleburtle.net/bob/hash/integer.html
     */
    a = ( a + 0x7ed55d16 ) + ( a << 12 );
    a = ( a ^ 0xc761c23c ) ^ ( a >> 19 );
    a = ( a + 0x165667b1 ) + ( a << 5  );
    a = ( a + 0xd3a2646c ) ^ ( a << 9  );
    a = ( a + 0xfd7046c5 ) + ( a << 3  );
    a = ( a ^ 0xb55a4f09 ) ^ ( a >> 16 );
    return a;
}

__device__ uintCUDA IdxBondArray_d
(
    intCUDA const x,
    intCUDA const y,
    intCUDA const z
)
{
    return   ( x & 7 ) +
           ( ( y & 7 ) << 3 ) +
           ( ( z & 7 ) << 6 );
}

template< typename T >
__device__ __host__ bool isPowerOfTwo( T const & x )
{
    return ! ( x == 0 ) && ! ( x & ( x - 1 ) );
}

uint32_t UpdaterGPUScBFM_AB_Type::linearizeBoxVectorIndex
(
    uint32_t const & ix,
    uint32_t const & iy,
    uint32_t const & iz
)
{
    #ifdef NOMAGIC
        return ( ix % mBoxX ) +
               ( iy % mBoxY ) * mBoxX +
               ( iz % mBoxZ ) * mBoxX * mBoxY;
    #else
        assert( isPowerOfTwo( mBoxXM1 + 1 ) );
        assert( isPowerOfTwo( mBoxYM1 + 1 ) );
        assert( isPowerOfTwo( mBoxZM1 + 1 ) );
        return   ( ix & mBoxXM1 ) +
               ( ( iy & mBoxYM1 ) << mBoxXLog2  ) +
               ( ( iz & mBoxZM1 ) << mBoxXYLog2 );
    #endif
}

__device__ uint32_t linearizeBoxVectorIndex
(
    uint32_t const & ix,
    uint32_t const & iy,
    uint32_t const & iz
)
{
    #if DEBUG_UPDATERGPUSCBFM_AB_TYPE > 10
        assert( isPowerOfTwo( mBoxXM1 + 1 ) );
        assert( isPowerOfTwo( mBoxYM1 + 1 ) );
        assert( isPowerOfTwo( mBoxZM1 + 1 ) );
    #endif
    return   ( ix & dcBoxXM1 ) +
           ( ( iy & dcBoxYM1 ) << dcBoxXLog2  ) +
           ( ( iz & dcBoxZM1 ) << dcBoxXYLog2 );
}

__device__ inline bool checkLattice
(
    hipTextureObject_t const texLattice,
    intCUDA  const x0,
    intCUDA  const y0,
    intCUDA  const z0,
    intCUDA  const dx,
    intCUDA  const dy,
    intCUDA  const dz,
    uintCUDA const axis
)
{
    uint8_t test = 0;
#if 0 // defined( NOMAGIC ) // boh versions successfully tested :)
    /* positions after movement. Why 2 times dx ??? */
    uint32_t const x1 = ( x0 + dx + dx ) & dcBoxXM1;
    uint32_t const y1 = ( y0 + dy + dy ) & dcBoxYM1;
    uint32_t const z1 = ( z0 + dz + dz ) & dcBoxZM1;
    switch ( axis )
    {
        #define TMP_FETCH( x,y,z ) \
            tex1Dfetch< uint8_t >( texLattice, linearizeBoxVectorIndex(x,y,z) )
        case 0: //-+x
            test = TMP_FETCH( x1, y0 - 1, z0     ) |
                   TMP_FETCH( x1, y0    , z0     ) |
                   TMP_FETCH( x1, y0 + 1, z0     ) |
                   TMP_FETCH( x1, y0 - 1, z0 - 1 ) |
                   TMP_FETCH( x1, y0    , z0 - 1 ) |
                   TMP_FETCH( x1, y0 + 1, z0 - 1 ) |
                   TMP_FETCH( x1, y0 - 1, z0 + 1 ) |
                   TMP_FETCH( x1, y0    , z0 + 1 ) |
                   TMP_FETCH( x1, y0 + 1, z0 + 1 );
            break;
        case 1: //-+y
            test = TMP_FETCH( x0 - 1, y1, z0 - 1 ) |
                   TMP_FETCH( x0    , y1, z0 - 1 ) |
                   TMP_FETCH( x0 + 1, y1, z0 - 1 ) |
                   TMP_FETCH( x0 - 1, y1, z0     ) |
                   TMP_FETCH( x0    , y1, z0     ) |
                   TMP_FETCH( x0 + 1, y1, z0     ) |
                   TMP_FETCH( x0 - 1, y1, z0 + 1 ) |
                   TMP_FETCH( x0    , y1, z0 + 1 ) |
                   TMP_FETCH( x0 + 1, y1, z0 + 1 );
            break;
        case 2: //-+z
            test = TMP_FETCH( x0 - 1, y0 - 1, z1 ) |
                   TMP_FETCH( x0    , y0 - 1, z1 ) |
                   TMP_FETCH( x0 + 1, y0 - 1, z1 ) |
                   TMP_FETCH( x0 - 1, y0    , z1 ) |
                   TMP_FETCH( x0    , y0    , z1 ) |
                   TMP_FETCH( x0 + 1, y0    , z1 ) |
                   TMP_FETCH( x0 - 1, y0 + 1, z1 ) |
                   TMP_FETCH( x0    , y0 + 1, z1 ) |
                   TMP_FETCH( x0 + 1, y0 + 1, z1 );
            break;
        #undef TMP_FETCH
    }
#else
    uint32_t const x1     =   ( x0 + dx + dx ) & dcBoxXM1;
    uint32_t const y1     = ( ( y0 + dy + dy ) & dcBoxYM1 ) << dcBoxXLog2;
    uint32_t const z1     = ( ( z0 + dz + dz ) & dcBoxZM1 ) << dcBoxXYLog2;
    uint32_t const x0Abs  =   ( x0     ) & dcBoxXM1;
    uint32_t const x0PDX  =   ( x0 + 1 ) & dcBoxXM1;
    uint32_t const x0MDX  =   ( x0 - 1 ) & dcBoxXM1;
    uint32_t const y0Abs  = ( ( y0     ) & dcBoxYM1 ) << dcBoxXLog2;
    uint32_t const y0PDY  = ( ( y0 + 1 ) & dcBoxYM1 ) << dcBoxXLog2;
    uint32_t const y0MDY  = ( ( y0 - 1 ) & dcBoxYM1 ) << dcBoxXLog2;
    uint32_t const z0Abs  = ( ( z0     ) & dcBoxZM1 ) << dcBoxXYLog2;
    uint32_t const z0PDZ  = ( ( z0 + 1 ) & dcBoxZM1 ) << dcBoxXYLog2;
    uint32_t const z0MDZ  = ( ( z0 - 1 ) & dcBoxZM1 ) << dcBoxXYLog2;

    switch ( axis )
    {
        case 0: //-+x
            test = tex1Dfetch< uint8_t >( texLattice, x1 + y0MDY + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0Abs + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0PDY + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0MDY + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0Abs + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0PDY + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0MDY + z0PDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0Abs + z0PDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x1 + y0PDY + z0PDZ );
            break;
        case 1: //-+y
            test = tex1Dfetch< uint8_t >( texLattice, x0MDX + y1 + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y1 + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y1 + z0MDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x0MDX + y1 + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y1 + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y1 + z0Abs ) |
                   tex1Dfetch< uint8_t >( texLattice, x0MDX + y1 + z0PDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y1 + z0PDZ ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y1 + z0PDZ );
            break;
        case 2: //-+z
            test = tex1Dfetch< uint8_t >( texLattice, x0MDX + y0MDY + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y0MDY + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y0MDY + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0MDX + y0Abs + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y0Abs + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y0Abs + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0MDX + y0PDY + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0Abs + y0PDY + z1 ) |
                   tex1Dfetch< uint8_t >( texLattice, x0PDX + y0PDY + z1 );
            break;
    }
#endif
    return test;
}

/**
 * @param[in] rn a random number used as a kind of seed for the RNG
 * @param[in] nMonomers number of max. monomers to work on, this is for
 *            filtering out excessive threads and was prior a __constant__
 *            But it is only used one(!) time in the kernel so the caching
 *            of constant memory might not even be used.
 *            @see https://web.archive.org/web/20140612185804/http://www.pixel.io/blog/2013/5/9/kernel-arguments-vs-__constant__-variables.html
 *            -> Kernel arguments are even put into constant memory it seems:
 *            @see "Section E.2.5.2 Function Parameters" in the "CUDA 5.5 C Programming Guide"
 *            __global__ function parameters are passed to the device:
 *             - via shared memory and are limited to 256 bytes on devices of compute capability 1.x,
 *             - via constant memory and are limited to 4 KB on devices of compute capability 2.x and higher.
 *            __device__ and __global__ functions cannot have a variable number of arguments.
 * Note: all of the three kernels do quite few work. They basically just fetch
 *       data, and check one condition and write out again. There isn't even
 *       a loop and most of the work seems to be boiler plate initialization
 *       code which could be cut if the kernels could be merged together.
 *       Why are there three kernels instead of just one
 *        -> for global synchronization
 */
__global__ void kernelSimulationScBFMCheckSpezies
(
    intCUDA           * const mPolymerSystem_d ,
    uint8_t           * const mLatticeTmp_d    ,
    MonoInfo          * const MonoInfo_d       ,
    hipTextureObject_t const texSpeciesIndices,
    uint32_t            const nMonomers        ,
    uint32_t            const rn               ,
    hipTextureObject_t const texLatticeRefOut
)
{
    int linId = blockIdx.x * blockDim.x + threadIdx.x;
    /* might be more readable to just return if the thread is masked ???
     * if ( ! ( linId < nMonomers ) )
     *     return;
     * I think it only works on newer CUDA versions ??? else the whole warp
     * might quit???
     */
    if ( linId < nMonomers )
    {
        // "select random monomer" ??? I don't see why this is random? texSpeciesIndices is not randomized!
        uint32_t const iMonomer   = tex1Dfetch< uint32_t >( texSpeciesIndices, linId );
        /* isn't this basically an array of structs where a struct of arrays
         * should be faster ??? */
        intCUDA  const x0         = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+0 );
        intCUDA  const y0         = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+1 );
        intCUDA  const z0         = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+2 );
        intCUDA  const properties = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+3 );

        //select random direction. Own implementation of an rng :S? But I think it at least# was initialized using the LeMonADE RNG ...
        uintCUDA const random_int = hash( hash( linId ) ^ rn ) % 6;

         //select random direction. !!! That table is kinda magic. there might be a better way ... E.g. using bitmasking. Also, what is with 0 in one direction ??? There is no way to e.g. get (0,1,-1) ... ???
         //0:-x; 1:+x; 2:-y; 3:+y; 4:-z; 5+z
        intCUDA const dx = DXTable_d[ random_int ];
        intCUDA const dy = DYTable_d[ random_int ];
        intCUDA const dz = DZTable_d[ random_int ];

#ifdef NONPERIODICITY
       /* check whether the new location of the particle would be inside the box
        * if the box is not periodic, if not, then don't move the particle */
        if ( ! ( 0 <= x0 + dx && x0 + dx < dcBoxXM1 &&
                 0 <= y0 + dy && y0 + dy < dcBoxYM1 &&
                 0 <= z0 + dz && z0 + dz < dcBoxZM1 ) )
        {
            return;
        }
#endif
        const unsigned nextNeigborSize = ( properties & 224 ) >> 5; // 224 = 0b1110 0000
        for ( unsigned u = 0; u < nextNeigborSize; ++u )
        {
            intCUDA const nN_X = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*MonoInfo_d[iMonomer].bondsMonomerIdx[u]+0 );
            intCUDA const nN_Y = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*MonoInfo_d[iMonomer].bondsMonomerIdx[u]+1 );
            intCUDA const nN_Z = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*MonoInfo_d[iMonomer].bondsMonomerIdx[u]+2 );
            if ( dpForbiddenBonds[ IdxBondArray_d( nN_X - x0 - dx, nN_Y - y0 - dy, nN_Z - z0 - dz ) ] )
                return;
        }

        if ( checkLattice( texLatticeRefOut, x0, y0, z0, dx, dy, dz, random_int >> 1 ) )
            return;

        // everything fits -> perform the move - add the information
        // possible move
        /* ??? can I simply & dcBoxXM1 ? this looks like something like
         * ( x0+dx ) % xmax is trying to be achieved. Using bitmasking for that
         * is only possible if dcBoxXM1+1 is a power of two ... */
        mPolymerSystem_d[ 4*iMonomer+3 ] = properties | ((random_int<<2)+1);
        mLatticeTmp_d[ linearizeBoxVectorIndex( x0+dx, y0+dy, z0+dz ) ] = 1;
    }
}

__global__ void kernelSimulationScBFMPerformSpecies
(
    intCUDA             * const mPolymerSystem_d ,
    uint8_t             * const mLattice_d       ,
    hipTextureObject_t   const texSpeciesIndices,
    uint32_t              const nMonomers        ,
    hipTextureObject_t   const texLatticeTmpRef
)
{
    int const linId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( linId < nMonomers )
    {
        uint32_t const iMonomer   = tex1Dfetch< uint32_t >( texSpeciesIndices, linId );
        intCUDA  const properties = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+3 );
        if ( properties & 1 != 0 )    // possible move
        {
            intCUDA  const x0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+0 );
            intCUDA  const y0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+1 );
            intCUDA  const z0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+2 );
            uintCUDA const random_int = ( properties & 28 ) >> 2; // 28 == 0b11100

            intCUDA const dx = DXTable_d[ random_int ];
            intCUDA const dy = DYTable_d[ random_int ];
            intCUDA const dz = DZTable_d[ random_int ];

            if ( checkLattice( texLatticeTmpRef, x0, y0, z0, dx, dy, dz, random_int >> 1 ) )
                return;

            // everything fits -> perform the move - add the information
            //mPolymerSystem_d[ 4*iMonomer+0 ] = x0 + dx;
            //mPolymerSystem_d[ 4*iMonomer+1 ] = y0 + dy;
            //mPolymerSystem_d[ 4*iMonomer+2 ] = z0 + dz;
            mPolymerSystem_d[ 4*iMonomer+3 ] = properties | 2; // indicating allowed move
            mLattice_d[ linearizeBoxVectorIndex( x0+dx, y0+dy, z0+dz ) ] = 1;
            mLattice_d[ linearizeBoxVectorIndex( x0, y0, z0 ) ] = 0;
        }
    }
}

__global__ void kernelSimulationScBFMZeroArraySpecies
(
    intCUDA             * const mPolymerSystem_d ,
    uint8_t             * const mLatticeTmp_d    ,
    hipTextureObject_t   const texSpeciesIndices,
    uint32_t              const nMonomers
)
{
    int linId = blockIdx.x * blockDim.x + threadIdx.x;
    if ( linId < nMonomers )
    {
        uint32_t const iMonomer = tex1Dfetch< uint32_t >( texSpeciesIndices, linId );
        intCUDA  const properties = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+3 );

        if ( ( properties & 3 ) != 0 )    //possible move
        {
            intCUDA const x0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+0 );
            intCUDA const y0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+1 );
            intCUDA const z0 = tex1Dfetch( texPolymerAndMonomerIsEvenAndOnXRef, 4*iMonomer+2 );

            //select random direction
            uintCUDA const random_int = ( properties & 28 ) >> 2;

            //0:-x; 1:+x; 2:-y; 3:+y; 4:-z; 5+z
            intCUDA const dx = DXTable_d[ random_int ];
            intCUDA const dy = DYTable_d[ random_int ];
            intCUDA const dz = DZTable_d[ random_int ];

            // possible move but not allowed
            if ( ( properties & 3 ) == 1 )
            {
                mLatticeTmp_d[ linearizeBoxVectorIndex( x0+dx, y0+dy, z0+dz ) ] = 0;
                mPolymerSystem_d[ 4*iMonomer+3 ] = properties & MASK5BITS; // delete the first 5 bits
            }
            else //allowed move with all circumstance
            {
                mPolymerSystem_d[ 4*iMonomer+0 ] = x0 + dx;
                mPolymerSystem_d[ 4*iMonomer+1 ] = y0 + dy;
                mPolymerSystem_d[ 4*iMonomer+2 ] = z0 + dz;
                mPolymerSystem_d[ 4*iMonomer+3 ] = properties & MASK5BITS; // delete the first 5 bits
                mLatticeTmp_d[ linearizeBoxVectorIndex( x0+dx, y0+dy, z0+dz ) ] = 0;
            }
            // everything fits -> perform the move - add the information
            //  mPolymerSystem_d[4*iMonomer+3] = properties & MASK5BITS; // delete the first 5 bits <- this comment was only for species B
        }
    }
}

UpdaterGPUScBFM_AB_Type::~UpdaterGPUScBFM_AB_Type()
{
    std::cout << "[" << __FILENAME__ << "::~UpdaterGPUScBFM_AB_Type" << "] destructor" << std::endl;

    delete[] mLattice;
    delete[] mPolymerSystem;
    delete[] mAttributeSystem;
    for ( size_t i = 0; i < nAllMonomers; ++i )
        delete monosNNidx[i];
    delete monosNNidx;

}

void UpdaterGPUScBFM_AB_Type::initialize( int iGpuToUse )
{
    /**** Print some GPU information ****/
    hipDeviceProp_t prop;

    int nGpus;
    CUDA_CHECK( hipGetDeviceCount( &nGpus ) );

    for ( int i = 0; i < nGpus; ++i )
    {
        CUDA_CHECK( hipGetDeviceProperties( &prop, i ) );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap: %s\n", prop.deviceOverlap ? "Enabled" : "Disabled" );
        printf( "Kernel execution timeout : %s\n", prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled" );
        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n", prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                prop.maxThreadsDim[0],
                prop.maxThreadsDim[1],
                prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                prop.maxGridSize[0],
                prop.maxGridSize[1],
                prop.maxGridSize[2] );
        printf( "\n" );
    }

    if ( iGpuToUse >= nGpus )
    {
        std::cout << "GPU with ID " << iGpuToUse << " not present. Only " << nGpus << " GPUs are available. Exiting..." << std::endl;
        throw std::runtime_error( "Can not find GPU or GPU not present. Exiting..." );
    }

    /* choose GPU to use */
    CUDA_CHECK( hipSetDevice( iGpuToUse ));


    /**** create the BondTable and copy to constant memory ****/
    bool * tmpForbiddenBonds = (bool *) malloc(sizeof(bool)*512);
    uint nAllowedBonds = 0;
    for(int i = 0; i < 512; i++)
    {
        tmpForbiddenBonds[i] = false;
        tmpForbiddenBonds[i] = mForbiddenBonds[i];
        if ( ! tmpForbiddenBonds[i] )
            nAllowedBonds++;
    }
    std::cout << "used bonds in simulation: " << nAllowedBonds << " / 108 " << std::endl;
    if ( nAllowedBonds != 108 )
    {
        std::stringstream msg;
        msg << "Wrong bond-set! Expected 108 allowed bonds, but got " << nAllowedBonds << ". Exiting...\n";
        throw std::runtime_error( msg.str() );
    }
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dpForbiddenBonds), tmpForbiddenBonds, sizeof(bool)*512 ) );
    free(tmpForbiddenBonds);

    /* create a table mapping the random int to directions whereto move the
     * monomers */
    std::cout << "copy DXYZTable: " << std::endl;
    intCUDA tmp_DXTable[6] = { -1,1,  0,0,  0,0 };
    intCUDA tmp_DYTable[6] = {  0,0, -1,1,  0,0 };
    intCUDA tmp_DZTable[6] = {  0,0,  0,0, -1,1 };
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( DXTable_d), tmp_DXTable, sizeof( intCUDA ) * 6 ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( DYTable_d), tmp_DYTable, sizeof( intCUDA ) * 6 ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( DZTable_d), tmp_DZTable, sizeof( intCUDA ) * 6 ) );

    /***************************creating look-up for species*****************************************/

    /* count monomers per species before allocating per species arrays */
    uint32_t * pMonomerSpecies = (uint32_t *) malloc( nAllMonomers * sizeof(uint32_t) );
    nMonomersSpeciesA = 0;
    nMonomersSpeciesB = 0;
    for ( uint32_t i = 0; i < nAllMonomers; ++i )
    {
        // monomer is odd or even / A or B
        if ( mAttributeSystem[i] == 1 )
        {
            nMonomersSpeciesA++;
            pMonomerSpecies[i] = 1;
        }
        else if ( mAttributeSystem[i] == 2 )
        {
            nMonomersSpeciesB++;
            pMonomerSpecies[i] = 2;
        }
        else
            throw std::runtime_error( "wrong attributes!!! Exiting... \n" );
    }
    std::cout << "nMonomersSpezies_A: " << nMonomersSpeciesA << std::endl;
    std::cout << "nMonomersSpezies_B: " << nMonomersSpeciesB << std::endl;
    if ( nMonomersSpeciesA + nMonomersSpeciesB != nAllMonomers )
        throw std::runtime_error( "Nr Of MonomerSpezies does not add up! Exiting... \n");

    MonomersSpeziesIdx_A_host = (uint32_t *) malloc( nMonomersSpeciesA * sizeof(uint32_t) );
    MonomersSpeziesIdx_B_host = (uint32_t *) malloc( nMonomersSpeciesB * sizeof(uint32_t) );

    /* sort monomers (their indices) into corresponding species array  */
    uint32_t nMonomersWrittenA = 0;
    uint32_t nMonomersWrittenB = 0;
    for ( uint32_t i = 0; i < nAllMonomers; ++i )
    {
        if ( pMonomerSpecies[i] == 1 )
            MonomersSpeziesIdx_A_host[ nMonomersWrittenA++ ] = i;
        else if ( pMonomerSpecies[i] == 2 )
            MonomersSpeziesIdx_B_host[ nMonomersWrittenB++ ] = i;
    }
    if ( nMonomersSpeciesA != nMonomersWrittenA )
        throw std::runtime_error( "Number of monomers copeid for species A does not add up! Exiting... \n" );
    if ( nMonomersSpeciesB != nMonomersWrittenB )
        throw std::runtime_error( "Number of monomers copeid for species B does not add up! Exiting... \n" );

    /* move species tables to GPU */
    CUDA_CHECK( hipMalloc((void **) &MonomersSpeziesIdx_A_device, (nMonomersSpeciesA)*sizeof(uint32_t)) );
    CUDA_CHECK( hipMalloc((void **) &MonomersSpeziesIdx_B_device, (nMonomersSpeciesB)*sizeof(uint32_t)) );
    CUDA_CHECK( hipMemcpy( MonomersSpeziesIdx_A_device, MonomersSpeziesIdx_A_host, (nMonomersSpeciesA)*sizeof(uint32_t), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy( MonomersSpeziesIdx_B_device, MonomersSpeziesIdx_B_host, (nMonomersSpeciesB)*sizeof(uint32_t), hipMemcpyHostToDevice) );

    /************************end: creating look-up for species*****************************************/

    /* calculate kernel config */
    /* ceilDiv better ??? */
    numblocksSpecies_A = (nMonomersSpeciesA-1)/NUMTHREADS+1;
    numblocksSpecies_B = (nMonomersSpeciesB-1)/NUMTHREADS+1;

    /****************************copy monomer informations ********************************************/
    mPolymerSystem_host =(intCUDA *) malloc((4*nAllMonomers+1)*sizeof(intCUDA));
    std::cout << "try to allocate : " << ((4*nAllMonomers+1)*sizeof(intCUDA)) << " bytes = " << ((4*nAllMonomers+1)*sizeof(intCUDA)/(1024.0)) << " kB = " << ((4*nAllMonomers+1)*sizeof(intCUDA)/(1024.0*1024.0)) << " MB coordinates on GPU " << std::endl;

    /* copy [ x0,y0,z0, x1 ... ] -> [ x0,y0,z0,p0, x1 ...]. Might be
     * an idea to use hipMemcpy2D to transfer this strided array to GPU.
     * At least for copying back the results, see below, but for this the
     * property field actually will be set in the next few lines */
    CUDA_CHECK( hipMalloc( (void **) &mPolymerSystem_device, ( 4*nAllMonomers+1 ) * sizeof( intCUDA ) ) );
    for ( uint32_t i =0; i < nAllMonomers; ++i )
    {
        mPolymerSystem_host[ 4*i+0 ] = (intCUDA) mPolymerSystem[ 3*i+0 ];
        mPolymerSystem_host[ 4*i+1 ] = (intCUDA) mPolymerSystem[ 3*i+1 ];
        mPolymerSystem_host[ 4*i+2 ] = (intCUDA) mPolymerSystem[ 3*i+2 ];
        mPolymerSystem_host[ 4*i+3 ] = 0;
    }

    // prepare and copy the connectivity matrix to GPU
    // the index on GPU starts at 0 and is one less than loaded
    int sizeMonoInfo = nAllMonomers * sizeof( MonoInfo );

    std::cout << "size of struct MonoInfo: " << sizeof(MonoInfo) << " bytes = " << (sizeof(MonoInfo)/(1024.0)) <<  "kB for one monomer connectivity " << std::endl;

    std::cout << "try to allocate : " << (sizeMonoInfo) << " bytes = " << (sizeMonoInfo/(1024.0)) <<  "kB = " << (sizeMonoInfo/(1024.0*1024.0)) <<  "MB for connectivity matrix on GPU " << std::endl;


    MonoInfo_host=(MonoInfo*) calloc(nAllMonomers,sizeof(MonoInfo));
    CUDA_CHECK(  hipMalloc((void **) &MonoInfo_device, sizeMonoInfo));   // Allocate array of structure on device


    for ( uint32_t i = 0; i < nAllMonomers; ++i )
    {
        //MonoInfo_host[i].size = monosNNidx[i]->size;
        if((monosNNidx[i]->size) > 7)
        {
            std::cout << "this GPU-model allows max 7 next neighbors but size is " << (monosNNidx[i]->size) << ". Exiting..." << std::endl;
            throw std::runtime_error( "Limit of connectivity on GPU reached! Exiting...\n" );
        }

        mPolymerSystem_host[4*i+3] |= ((intCUDA)(monosNNidx[i]->size)) << 5;
        //cout << "mono:" << i << " vs " << (i) << endl;
        //cout << "numElements:" << MonoInfo_host[i].size << " vs " << monosNNidx[i]->size << endl;

        for(unsigned u=0; u < MAX_CONNECTIVITY; u++)
        {
            MonoInfo_host[i].bondsMonomerIdx[u] = monosNNidx[i]->bondsMonomerIdx[u];

            //cout << "bond["<< u << "]: " << MonoInfo_host[i].bondsMonomerIdx[u] << " vs " << monosNNidx[i]->bondsMonomerIdx[u] << endl;
        }
    }
    CUDA_CHECK( hipMemcpy( MonoInfo_device, MonoInfo_host, sizeMonoInfo, hipMemcpyHostToDevice ) );

    checkSystem();

    /* creating lattice */
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXM1   ), &mBoxXM1   , sizeof( mBoxXM1    ) ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxYM1   ), &mBoxYM1   , sizeof( mBoxYM1    ) ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZM1   ), &mBoxZM1   , sizeof( mBoxZM1    ) ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXLog2 ), &mBoxXLog2 , sizeof( mBoxXLog2  ) ) );
    CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXYLog2), &mBoxXYLog2, sizeof( mBoxXYLog2 ) ) );

    mLatticeOut_host = (uint8_t *) malloc( mBoxX*mBoxY*mBoxZ*sizeof(uint8_t));
    mLatticeTmp_host = (uint8_t *) malloc( mBoxX*mBoxY*mBoxZ*sizeof(uint8_t));
    std::cout << "try to allocate : " << (mBoxX*mBoxY*mBoxZ*sizeof(uint8_t)) << " bytes = " << (mBoxX*mBoxY*mBoxZ*sizeof(uint8_t)/(1024.0*1024.0)) << " MB lattice on GPU " << std::endl;
    CUDA_CHECK( hipMalloc( (void **) &mLatticeOut_device, mBoxX * mBoxY * mBoxZ * sizeof( *mLatticeOut_device ) ) );
    CUDA_CHECK( hipMalloc( (void **) &mLatticeTmp_device, mBoxX * mBoxY * mBoxZ * sizeof( *mLatticeTmp_device ) ) );
    CUDA_CHECK( hipMemset( mLatticeTmp_device, 0, mBoxX * mBoxY * mBoxZ * sizeof( *mLatticeTmp_device ) ) );

    std::memset( mLatticeOut_host, 0, mBoxX * mBoxY * mBoxZ * sizeof( *mLatticeOut_host ) );
    for ( int t = 0; t < nAllMonomers; ++t )
    {
        #ifdef USEZCURVE
            uint32_t xk = mPolymerSystem[ 3*t+0 ] & mBoxXM1;
            uint32_t yk = mPolymerSystem[ 3*t+1 ] & mBoxYM1;
            uint32_t zk = mPolymerSystem[ 3*t+2 ] & mBoxZM1;
            uint32_t inter3 = interleave3( xk/2 , yk/2, zk/2 );
            mLatticeOut_host[ ( ( mPolymerSystem_host[ 4*t+3 ] & 1 ) << 23 ) + inter3 ] = 1;
        #else
        mLatticeOut_host[ linearizeBoxVectorIndex( mPolymerSystem[ 3*t+0 ],
                                                   mPolymerSystem[ 3*t+1 ],
                                                   mPolymerSystem[ 3*t+2 ] ) ] = 1;
        #endif
    }
    CUDA_CHECK( hipMemcpy( mLatticeOut_device, mLatticeOut_host, mBoxX * mBoxY * mBoxZ * sizeof( *mLatticeOut_host ), hipMemcpyHostToDevice ) );
    CUDA_CHECK( hipMemcpy( mPolymerSystem_device, mPolymerSystem_host, ( 4*nAllMonomers+1 ) * sizeof( intCUDA ), hipMemcpyHostToDevice ) );

    /* bind textures */
    hipBindTexture( 0, texPolymerAndMonomerIsEvenAndOnXRef, mPolymerSystem_device, ( 4*nAllMonomers+1 ) * sizeof( intCUDA ) );

    /* new with texture object... they said it would be easier -.- */
    hipResourceDesc resDescA;
    memset( &resDescA, 0, sizeof( resDescA ) );
    resDescA.resType                = hipResourceTypeLinear;
    resDescA.res.linear.desc.f      = hipChannelFormatKindUnsigned;
    resDescA.res.linear.desc.x      = 32; // bits per channel
    hipResourceDesc resDescB = resDescA;
    hipResourceDesc resDescRefOut = resDescA;
    resDescA.res.linear.devPtr      = MonomersSpeziesIdx_A_device;
    resDescA.res.linear.sizeInBytes = nMonomersSpeciesA * sizeof( uint32_t );
    resDescB.res.linear.devPtr      = MonomersSpeziesIdx_B_device;
    resDescB.res.linear.sizeInBytes = nMonomersSpeciesB * sizeof( uint32_t );

    hipTextureDesc texDescROM;
    memset( &texDescROM, 0, sizeof( texDescROM ) );
    texDescROM.readMode = hipReadModeElementType;

    /* the last three arguments are pointers to constants! */
    hipCreateTextureObject( &texSpeciesIndicesA, &resDescA, &texDescROM, NULL );
    hipCreateTextureObject( &texSpeciesIndicesB, &resDescB, &texDescROM, NULL );

    /* lattice textures */
    resDescRefOut.res.linear.desc.x = 8; // bits per channel
    resDescRefOut.res.linear.sizeInBytes = mBoxX*mBoxY*mBoxZ*sizeof(uint8_t);
    hipResourceDesc resDescTmpRef = resDescRefOut;
    resDescRefOut.res.linear.devPtr = mLatticeOut_device;
    resDescTmpRef.res.linear.devPtr = mLatticeTmp_device;

    hipCreateTextureObject( &texLatticeRefOut, &resDescRefOut, &texDescROM, NULL );
    hipCreateTextureObject( &texLatticeTmpRef, &resDescTmpRef, &texDescROM, NULL );

    /* The memory transfer is this complex because int16_t needs to be
     * converted to int32_t if cudaInt == int16_t */
    if ( sizeof( *mPolymerSystem ) == sizeof( *mPolymerSystem_device ) )
    {
        std::cerr << "[" << __FILENAME__ << "::initialize] Can use hipMemcpy2D (untested!)\n";
        throw std::runtime_error( "Untested. Uncomment if you work on it." );
        CUDA_CHECK( hipMemcpy2D(
            mPolymerSystem       , 3 * sizeof( *mPolymerSystem ),
            mPolymerSystem_device, 4 * sizeof( *mPolymerSystem_device ),
            3 * sizeof( *mPolymerSystem ), nAllMonomers,
            hipMemcpyDeviceToHost
        ) );
    }
    else
    {
        std::cerr << "[" << __FILENAME__ << "::initialize] Can't use hipMemcpy2D"
            << " ( sizeof polymersystem, host: " << sizeof( *mPolymerSystem )
            << ", GPU: " << sizeof( *mPolymerSystem_device ) << ")\n";
        CUDA_CHECK( hipMemcpy( mPolymerSystem_host, mPolymerSystem_device, ( 4*nAllMonomers+1 ) * sizeof( intCUDA ), hipMemcpyDeviceToHost ) );
        for( uint32_t i = 0; i < nAllMonomers; ++i )
        {
            mPolymerSystem[ 3*i+0 ] = (int32_t) mPolymerSystem_host[ 4*i+0 ];
            mPolymerSystem[ 3*i+1 ] = (int32_t) mPolymerSystem_host[ 4*i+1 ];
            mPolymerSystem[ 3*i+2 ] = (int32_t) mPolymerSystem_host[ 4*i+2 ];
        }
    }

    std::cout << "check system before simulation: " << std::endl;
    checkSystem();
}

/**
 * !!! Problems:
 *  Note that this simply bitmasks negative values, e.g. x=-4 = 0xfffffffc
 *  Note that 0xfc = 1111 1100b and &7 -> 100. Vice-versa 4 = 100b ...
 *   => this clashes !!! As a simple runtime test shows, both are indeed used!
 */
int UpdaterGPUScBFM_AB_Type::IndexBondArray( int const x, int const y, int const z )
{
#ifndef NDEBUG2
    if ( x == -4 || x == 4 )
    {
        /* Found negative x=-4 = fffffffc */
        std::cout << "[" << __FILENAME__ << "::IndexBondArray] +-4 x="
                  << x << " = " << std::hex << x << std::dec << std::endl;
    }
#endif
    /* 7 == 0b111, i.e. truncate the lowest 3 bits */
    return   ( x & 7 ) +
           ( ( y & 7 ) << 3 ) +
           ( ( z & 7 ) << 6 );
}

void UpdaterGPUScBFM_AB_Type::setNrOfAllMonomers( uint32_t rnAllMonomers )
{
    nAllMonomers = rnAllMonomers;
    std::cout << "[" << __FILENAME__ << "::setNrOfAllMonomers" << "] used monomers in simulation: " << nAllMonomers << std::endl;

    mAttributeSystem = new int32_t[nAllMonomers];
    mPolymerSystem   = new int32_t[nAllMonomers*3+1];    /* why +1 ??? */

    //idx is reduced by one compared to the file
    monosNNidx = new MonoNNIndex*[nAllMonomers];
    for ( uint32_t a = 0; a < nAllMonomers; ++a )
    {
        monosNNidx[a] = new MonoNNIndex();
        monosNNidx[a]->size=0;
        for ( unsigned o = 0; o < MAX_CONNECTIVITY; ++o )
            monosNNidx[a]->bondsMonomerIdx[o]=0;
    }
}

void UpdaterGPUScBFM_AB_Type::setPeriodicity(bool isPeriodicX, bool isPeriodicY, bool isPeriodicZ)
{
    //check if we are using periodic boundary condition and the simulations are do so
#ifdef NONPERIODICITY
    if((isPeriodicX == true) || (isPeriodicY == true) || (isPeriodicZ == true) )
    {
        std::stringstream errormessage;
        errormessage<<"Simulation is intended to use NON-PERIODIC BOUNDARY conditions.\n";
        errormessage<<"But in BFM-File the PERIODICITY is set to:\n";
        errormessage<<"In X:"<<isPeriodicX<<"\n";
        errormessage<<"In Y:"<<isPeriodicY<<"\n";
        errormessage<<"In Z:"<<isPeriodicZ<<"\n";
        errormessage<<"Logical Error! Exiting...\n";
        throw std::runtime_error(errormessage.str());
    }
#else
    if((isPeriodicX == false) || (isPeriodicY == false) || (isPeriodicZ == false) )
    {
        std::stringstream errormessage;
        errormessage<<"Simulation is intended to use PERIODIC BOUNDARY conditions.\n";
        errormessage<<"But in BFM-File the PERIODICITY is set to:\n";
        errormessage<<"In X:"<<isPeriodicX<<"\n";
        errormessage<<"In Y:"<<isPeriodicY<<"\n";
        errormessage<<"In Z:"<<isPeriodicZ<<"\n";
        errormessage<<"Logical Error! Exiting...\n";
        throw std::runtime_error(errormessage.str());
    }
#endif

}

void UpdaterGPUScBFM_AB_Type::setNetworkIngredients( uint32_t numPEG, uint32_t numPEGArm, uint32_t numCL )
{
    nStars              = numPEG;    //number of Stars
    nMonomersPerStarArm = numPEGArm; //number OfMonomersPerStarArm
    nCrosslinker        = numCL;     //number of Crosslinker

    std::cout << "NumPEG on GPU         : " << nStars              << std::endl;
    std::cout << "NumPEGArmlength on GPU: " << nMonomersPerStarArm << std::endl;
    std::cout << "NumCrosslinker on GPU : " << nCrosslinker        << std::endl;

    //if (nMonomersPerStarArm != 29)
        //throw std::runtime_error("nMonomersPerStarArm should be 29! Exiting...\n");
    //if ((nMonomersPerStarArm%2) != 1)
        //    throw std::runtime_error("nMonomersPerStarArm should be an odd number! Exiting...\n");
}

void UpdaterGPUScBFM_AB_Type::setConnectivity(uint32_t monoidx1, uint32_t monoidx2)
{
    monosNNidx[monoidx1]->bondsMonomerIdx[monosNNidx[monoidx1]->size] = monoidx2;
    //monosNNidx[monoidx2]->bondsMonomerIdx[monosNNidx[monoidx2]->size] = monoidx1;

    monosNNidx[monoidx1]->size++;
    //monosNNidx[monoidx2]->size++;

    //if((monosNNidx[monoidx1]->size > MAX_CONNECTIVITY) || (monosNNidx[monoidx2]->size > MAX_CONNECTIVITY))
    if ( monosNNidx[monoidx1]->size > MAX_CONNECTIVITY )
        throw std::runtime_error("MAX_CONNECTIVITY  exceeded! Exiting...\n");
}

void UpdaterGPUScBFM_AB_Type::setLatticeSize
(
    uint32_t const boxX,
    uint32_t const boxY,
    uint32_t const boxZ
)
{
    mBoxX   = boxX;
    mBoxY   = boxY;
    mBoxZ   = boxZ;
    mBoxXM1 = boxX-1;
    mBoxYM1 = boxY-1;
    mBoxZM1 = boxZ-1;

    /* determine log2 for mBoxX and mBoxX * mBoxY to be used for bitshifting
     * the indice instead of multiplying ... WHY??? I don't think it is faster,
     * but much less readable */
    mBoxXLog2 = 0;
    uint32_t dummy = boxX;
    while ( dummy >>= 1 ) ++mBoxXLog2;
    mBoxXYLog2 = 0;
    dummy = boxX*boxY;
    while ( dummy >>= 1 ) ++mBoxXYLog2;

    std::cout
        << "use bit shift for boxX     : (1 << "<< mBoxXLog2  << " ) = "
        << ( 1 << mBoxXLog2  ) << " = " << mBoxX
        << "use bit shift for boxX*boxY: (1 << "<< mBoxXYLog2 << " ) = "
        << ( 1 << mBoxXYLog2 ) << " = " << mBoxX*boxY
        << std::endl;

    // check if shift is correct
    if ( boxX != ( 1 << mBoxXLog2 ) || boxX * boxY != ( 1 << mBoxXYLog2 ) )
        throw std::runtime_error( "Could not determine value for bit shift. Sure your box size is a power of 2? Exiting...\n" );

    //init lattice
    mLattice = new uint8_t[ mBoxX * mBoxY * mBoxZ ];
    std::memset( (void *) mLattice, 0, mBoxX * mBoxY * mBoxZ * sizeof( *mLattice ) );
}

void UpdaterGPUScBFM_AB_Type::populateLattice()
{
    std::memset( mLattice, 0, mBoxX * mBoxY * mBoxZ * sizeof( *mLattice ) );
    for ( size_t i = 0; i < nAllMonomers; ++i )
    {
        mLattice[ linearizeBoxVectorIndex( mPolymerSystem[3*i+0],
                                           mPolymerSystem[3*i+1],
                                           mPolymerSystem[3*i+2] ) ] = 1;
    }
}

/**
 * Checks for excluded volume condition and for correctness of all monomer bonds
 * Beware, it useses and thereby thrashes mLattice. Might be cleaner to declare
 * as const and malloc and free some temporary buffer, but the time ...
 * https://randomascii.wordpress.com/2014/12/10/hidden-costs-of-memory-allocation/
 * "In my tests, for sizes ranging from 8 MB to 32 MB, the cost for a new[]/delete[] pair averaged about 7.5 μs (microseconds), split into ~5.0 μs for the allocation and ~2.5 μs for the free."
 *  => ~40k cycles
 */
void UpdaterGPUScBFM_AB_Type::checkSystem()
{
    /**
     * Test for excluded volume by setting all lattice points and count the
     * toal lattice points occupied. If we have overlap this will be smaller
     * than calculated for zero overlap!
     * mPolymerSystem only stores the lower left front corner of the 2x2x2
     * monomer cube. Use that information to set all 8 cells in the lattice
     * to 'occupied'
     */
    /*
     Lattice is an array of size Box_X*Box_Y*Box_Z. PolymerSystem holds the monomer positions which I strongly guess are supposed to be in the range 0<=x<Box_X. If I see correctly, then this part checks for excluded volume by occupying a 2x2x2 cube for each monomer in Lattice and then counting the total occupied cells and compare it to the theoretical value of nMonomers * 8. But Lattice seems to be too small for that kinda usage! I.e. for two particles, one being at x=0 and the other being at x=Box_X-1 this test should return that the excluded volume condition is not met! Therefore the effective box size is actually (Box_X-1,Box_X-1,Box_Z-1) which in my opinion should be a bug ??? */
    std::memset( mLattice, 0, mBoxX * mBoxY * mBoxZ * sizeof( *mLattice ) );
    for ( int i = 0; i < nAllMonomers; ++i )
    {
        int32_t const & x = mPolymerSystem[ 3*i   ];
        int32_t const & y = mPolymerSystem[ 3*i+1 ];
        int32_t const & z = mPolymerSystem[ 3*i+2 ];
        /**
         * @verbatim
         *           ...+---+---+
         *     ...'''   | 6 | 7 |
         *    +---+---+ +---+---+    y
         *    | 2 | 3 | | 4 | 5 |    ^ z
         *    +---+---+ +---+---+    |/
         *    | 0 | 1 |   ...'''     +--> x
         *    +---+---+'''
         * @endverbatim
         */
        mLattice[ linearizeBoxVectorIndex( x  , y  , z   ) ] = 1; /* 0 */
        mLattice[ linearizeBoxVectorIndex( x+1, y  , z   ) ] = 1; /* 1 */
        mLattice[ linearizeBoxVectorIndex( x  , y+1, z   ) ] = 1; /* 2 */
        mLattice[ linearizeBoxVectorIndex( x+1, y+1, z   ) ] = 1; /* 3 */
        mLattice[ linearizeBoxVectorIndex( x  , y  , z+1 ) ] = 1; /* 4 */
        mLattice[ linearizeBoxVectorIndex( x+1, y  , z+1 ) ] = 1; /* 5 */
        mLattice[ linearizeBoxVectorIndex( x  , y+1, z+1 ) ] = 1; /* 6 */
        mLattice[ linearizeBoxVectorIndex( x+1, y+1, z+1 ) ] = 1; /* 7 */
    }
    /* check total occupied cells inside lattice to ensure that the above
     * transfer went without problems. Note that the number will be smaller
     * if some monomers overlap!
     * Could also simply reduce mLattice with +, I think, because it only
     * cotains 0 or 1 ??? */
    unsigned nOccupied = 0;
    for ( unsigned i = 0u; i < mBoxX * mBoxY * mBoxZ; ++i )
        nOccupied += mLattice[i] != 0;
    if ( ! ( nOccupied == nAllMonomers * 8 ) )
    {
        std::stringstream msg;
        msg << "[" << __FILENAME__ << "::~checkSystem" << "] "
            << "Occupation count in mLattice is wrong! Expected 8*nMonomers="
            << 8 * nAllMonomers << " occupied cells, but got " << nOccupied;
        throw std::runtime_error( msg.str() );
    }

    /**
     * Check bonds i.e. that |dx|<=3 and whether it is allowed by the given
     * bond set
     */
    for ( unsigned i = 0; i < nAllMonomers; ++i )
    for ( unsigned idxNN = 0; idxNN < monosNNidx[i]->size; ++idxNN )
    {
        /* calculate the bond vector between the neighbor and this particle
         * neighbor - particle = ( dx, dy, dz ) */
        int32_t const dx = mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+0 ] - mPolymerSystem[ 3*i+0 ];
        int32_t const dy = mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+1 ] - mPolymerSystem[ 3*i+1 ];
        int32_t const dz = mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+2 ] - mPolymerSystem[ 3*i+2 ];

        int erroneousAxis = -1;
        if ( ! ( -3 <= dx && dx <= 3 ) ) erroneousAxis = 0;
        if ( ! ( -3 <= dy && dy <= 3 ) ) erroneousAxis = 1;
        if ( ! ( -3 <= dz && dz <= 3 ) ) erroneousAxis = 2;
        if ( erroneousAxis >= 0 || mForbiddenBonds[ IndexBondArray( dx, dy, dz ) ] )
        {
            std::stringstream msg;
            msg << "[" << __FILENAME__ << "::checkSystem] ";
            if ( erroneousAxis > 0 )
                msg << "Invalid " << 'X' + erroneousAxis << "Bond: ";
            if ( mForbiddenBonds[ IndexBondArray( dx, dy, dz ) ] )
                msg << "This particular bond is forbidden: ";
            msg << "(" << dx << "," << dy<< "," << dz << ") between monomer "
                << i+1 << " at (" << mPolymerSystem[3*i+0] << ","
                                  << mPolymerSystem[3*i+1] << ","
                                  << mPolymerSystem[3*i+2] << ") and monomer "
                << monosNNidx[i]->bondsMonomerIdx[idxNN]+1 << " at ("
                << mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+0 ] << ","
                << mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+1 ] << ","
                << mPolymerSystem[ 3*monosNNidx[i]->bondsMonomerIdx[idxNN]+2 ] << ")"
                << std::endl;
             throw std::runtime_error( msg.str() );
        }
    }
}

void UpdaterGPUScBFM_AB_Type::runSimulationOnGPU
(
    int32_t const nMonteCarloSteps
)
{
    std::clock_t const t0 = std::clock();

    /* run simulation */
    for ( int32_t iStep = 1; iStep <= nMonteCarloSteps; ++iStep )
    {
        /* one Monte-Carlo step */
        for ( uint32_t iSubStep = 0; iSubStep < 2; ++iSubStep )
        {
            switch ( randomNumbers.r250_rand32() % 2 )
            {
                case 0:  // run Spezies_A monomers
                    kernelSimulationScBFMCheckSpezies
                    <<< numblocksSpecies_A, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeTmp_device,
                        MonoInfo_device, texSpeciesIndicesA,
                        nMonomersSpeciesA, randomNumbers.r250_rand32(),
                        texLatticeRefOut
                    );
                    kernelSimulationScBFMPerformSpecies
                    <<< numblocksSpecies_A, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeOut_device,
                        texSpeciesIndicesA, nMonomersSpeciesA,
                        texLatticeTmpRef
                    );
                    kernelSimulationScBFMZeroArraySpecies
                    <<< numblocksSpecies_A, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeTmp_device,
                        texSpeciesIndicesA, nMonomersSpeciesA
                    );
                    break;

                case 1: // run Spezies_B monomers
                    kernelSimulationScBFMCheckSpezies
                    <<< numblocksSpecies_B, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeTmp_device,
                        MonoInfo_device, texSpeciesIndicesB,
                        nMonomersSpeciesB, randomNumbers.r250_rand32(),
                        texLatticeRefOut
                    );
                    kernelSimulationScBFMPerformSpecies
                    <<< numblocksSpecies_B, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeOut_device,
                        texSpeciesIndicesB, nMonomersSpeciesB,
                        texLatticeTmpRef
                    );
                    kernelSimulationScBFMZeroArraySpecies
                    <<< numblocksSpecies_B, NUMTHREADS >>>(
                        mPolymerSystem_device, mLatticeTmp_device,
                        texSpeciesIndicesB, nMonomersSpeciesB
                    );
                    break;

                default: break;
            }
        }
    }

    /* all MCS are done- copy information back from GPU to host */
    CUDA_CHECK( hipMemcpy( mLatticeTmp_host, mLatticeTmp_device, mBoxX * mBoxY * mBoxZ * sizeof( uint8_t ), hipMemcpyDeviceToHost ) );

    unsigned nOccupied = 0;
    for ( unsigned i = 0u; i < mBoxX * mBoxY * mBoxZ; ++i )
        nOccupied += mLatticeTmp_host[i] != 0;
    if ( nOccupied != 0 )
    {
        std::stringstream msg;
        msg << "latticeTmp occupation (" << nOccupied << ") should be 0! Exiting ...\n";
        throw std::runtime_error( msg.str() );
    }

    /* why isn't this copied directly into mLattice ??? */
    CUDA_CHECK( hipMemcpy( mLatticeOut_host, mLatticeOut_device, mBoxX * mBoxY * mBoxZ * sizeof( uint8_t ), hipMemcpyDeviceToHost ) );
    for ( int i = 0; i < mBoxX * mBoxY * mBoxZ; ++i )
        mLattice[i] = mLatticeOut_host[i];

    //start -z-order
    /*
    cout << "save -- recalculate mLattice: " << endl;
    //fetch from device and check again
        for(int i = 0; i < LATTICE_X*LATTICE_Y*LATTICE_Z; i++)
        {
            if(mLatticeOut_host[i]==1)
            {
                uint32_t dummyhost = i;
                uint32_t onX = (dummyhost / (1 <<23)); //0 on O, 1 on X
                uint32_t zl = 2*( deinterleave3_Z((dummyhost % (1 <<23)))) + onX;
                uint32_t yl = 2*( deinterleave3_Y((dummyhost % (1 <<23)))) + onX;
                uint32_t xl = 2*( deinterleave3_X((dummyhost % (1 <<23)))) + onX;


                //cout << "X: " << xl << "\tY: " << yl << "\tZ: " << zl<< endl;
                mLattice[xl + (yl << LATTICE_XPRO) + (zl << LATTICE_PROXY)] = 1;

            }

        }
        //end -z-order
    */

    /* copy into mPolymerSystem and drop the property tag while doing so.
     * would be easier and probably more efficient if mPolymerSystem_device/host
     * would be a struct of arrays instead of an array of structs !!! */
    CUDA_CHECK( hipMemcpy( mPolymerSystem_host, mPolymerSystem_device, ( 4*nAllMonomers+1 ) * sizeof( intCUDA ), hipMemcpyDeviceToHost ) );
    for ( uint32_t i = 0; i < nAllMonomers; ++i )
    {
        mPolymerSystem[ 3*i+0 ] = (int32_t) mPolymerSystem_host[ 4*i+0 ];
        mPolymerSystem[ 3*i+1 ] = (int32_t) mPolymerSystem_host[ 4*i+1 ];
        mPolymerSystem[ 3*i+2 ] = (int32_t) mPolymerSystem_host[ 4*i+2 ];
    }

    checkSystem();

    std::clock_t const t1 = std::clock();
    double const dt = float(t1-t0) / CLOCKS_PER_SEC;
    std::cout
    << "run time (GPU): " << nMonteCarloSteps << "\n"
    << "mcs = " << nMonteCarloSteps  << "  speed [performed monomer try and move/s] = MCS*N/t: "
    << nMonteCarloSteps * ( nAllMonomers / dt )  << "     runtime[s]:" << dt << std::endl;
}

void UpdaterGPUScBFM_AB_Type::cleanup()
{
    // copy information from GPU to Host
    CUDA_CHECK( hipMemcpy( mLattice, mLatticeOut_device, mBoxX * mBoxY * mBoxZ * sizeof(uint8_t), hipMemcpyDeviceToHost ) );
    CUDA_CHECK( hipMemcpy( mPolymerSystem_host, mPolymerSystem_device, (4*nAllMonomers+1)*sizeof(intCUDA), hipMemcpyDeviceToHost ) );
    for ( uint32_t i= 0 ; i < nAllMonomers; ++i )
    {
        mPolymerSystem[ 3*i+0 ] = (int32_t) mPolymerSystem_host[ 4*i+0 ];
        mPolymerSystem[ 3*i+1 ] = (int32_t) mPolymerSystem_host[ 4*i+1 ];
        mPolymerSystem[ 3*i+2 ] = (int32_t) mPolymerSystem_host[ 4*i+2 ];
    }

    checkSystem();

    // copy connectivity matrix back from device to host
    int sizeMonoInfo = nAllMonomers * sizeof(MonoInfo);
    CUDA_CHECK( hipMemcpy(MonoInfo_host, MonoInfo_device, sizeMonoInfo, hipMemcpyDeviceToHost));
    for (uint32_t i=0; i<nAllMonomers; i++)
    {
        //if(MonoInfo_host[i].size != monosNNidx[i]->size)
        if (  ( ( mPolymerSystem_host[4*i+3] & 224 ) >> 5 ) != monosNNidx[i]->size )
        {
            std::cout << "connectivity error after simulation run" << std::endl;
            std::cout << "mono:" << i << " vs " << (i) << std::endl;
            //cout << "numElements:" << MonoInfo_host[i].size << " vs " << monosNNidx[i]->size << endl;
            std::cout << "numElements:" << ((mPolymerSystem_host[4*i+3]&224)>>5) << " vs " << monosNNidx[i]->size << std::endl;

            throw std::runtime_error("Connectivity is corrupted! Maybe your Simulation is wrong! Exiting...\n");
        }
        for ( unsigned u = 0; u < MAX_CONNECTIVITY; ++u )
        {
            if ( MonoInfo_host[i].bondsMonomerIdx[u] != monosNNidx[i]->bondsMonomerIdx[u] )
            {
                std::cout << "connectivity error after simulation run" << std::endl;
                std::cout << "mono:" << i << " vs " << (i) << std::endl;

                std::cout << "bond["<< u << "]: " << MonoInfo_host[i].bondsMonomerIdx[u] << " vs " << monosNNidx[i]->bondsMonomerIdx[u] << std::endl;

                throw std::runtime_error("Connectivity is corrupted! Maybe your Simulation is wrong! Exiting...\n");
            }
        }
    }
    std::cout << "no errors in connectivity matrix after simulation run" << std::endl;

    checkSystem();

    //unbind texture reference to free resource
    hipUnbindTexture( texPolymerAndMonomerIsEvenAndOnXRef );
    hipDestroyTextureObject( texSpeciesIndicesA );
    hipDestroyTextureObject( texSpeciesIndicesB );
    texSpeciesIndicesA = 0;
    texSpeciesIndicesB = 0;

    //free memory on GPU
    hipFree( mLatticeOut_device          );
    hipFree( mLatticeTmp_device          );
    hipFree( mPolymerSystem_device       );
    hipFree( MonoInfo_device             );
    hipFree( MonomersSpeziesIdx_A_device );
    hipFree( MonomersSpeziesIdx_B_device );

    //free memory on CPU
    free( mPolymerSystem_host       );
    free( MonoInfo_host             );
    free( mLatticeOut_host          );
    free( mLatticeTmp_host          );
    free( MonomersSpeziesIdx_A_host );
    free( MonomersSpeziesIdx_B_host );
}
